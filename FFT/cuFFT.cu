#include <iostream>  
#include <cmath>  
#include <vector>  
#include <complex>  
#include <hip/hip_runtime.h>  
#include <hipfft/hipfft.h>  
  
using namespace std;  
  
int main() {  
    int N = 8;  
    hipfftDoubleComplex *h_data = new hipfftDoubleComplex[N];  
    for (int i = 0; i < N; ++i) {  
        h_data[i].x = (i < 4) ? 1.0 : 0.0;  
        h_data[i].y = 0.0;  
    }  
  
    hipfftDoubleComplex *d_data;  
    hipMalloc((void**)&d_data, N * sizeof(hipfftDoubleComplex));  
    hipMemcpy(d_data, h_data, N * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);  
  
    hipfftHandle plan;  
    hipfftPlan1d(&plan, N, HIPFFT_Z2Z, 1);  
    hipfftExecZ2Z(plan, d_data, d_data, HIPFFT_FORWARD);  
    hipMemcpy(h_data, d_data, N * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);  
  
    cout << "FFT result:" << endl;  

    // 输出“实部+虚部”的形式
    for (int i = 0; i < N; ++i) {  
        cout << h_data[i].x << ", " << h_data[i].y << "*i" << endl;  
    }

    // 输出“幅值+相角”的形式
    /*
    for (int i = 0; i < N; ++i) {  
        double magnitude = sqrt(h_data[i].x * h_data[i].x + h_data[i].y * h_data[i].y);  
        double phase = atan2(h_data[i].y, h_data[i].x) / M_PI;  
        cout << magnitude << ", " << phase << " * PI" << endl;  
    }  
    */
  
    hipfftDestroy(plan);  
    hipFree(d_data);  
    delete[] h_data;  
    return 0;  
}  

// nvcc cuFFT.cu -lcufft -o cuFFT
// ./cuFFT
